#include "hip/hip_runtime.h"
#include "../kernels.hpp"

__global__ void combine(float4* __restrict out, const deviceFFT_t* __restrict d_x, const deviceFFT_t* __restrict d_y, const deviceFFT_t* __restrict d_z){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    deviceFFT_t x = __ldg(&d_x[idx]);
    deviceFFT_t y = __ldg(&d_y[idx]);
    deviceFFT_t z = __ldg(&d_z[idx]);

    float4 this_out;
    this_out.x = x.x;
    this_out.y = y.x;
    this_out.z = z.x;

    out[idx] = this_out;
}

__global__ void combine_parallel(float4* __restrict out, const deviceFFT_t* __restrict d_x, const deviceFFT_t* __restrict d_y, const deviceFFT_t* __restrict d_z, int3 local_grid_size, int3 local_coords, int overload, int nlocal, int ng){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= nlocal)return;
    deviceFFT_t x = __ldg(&d_x[idx]);
    deviceFFT_t y = __ldg(&d_y[idx]);
    deviceFFT_t z = __ldg(&d_z[idx]);

    int3 globalIdx3d = HACCGPM::parallel::get_global_index(idx,ng,local_grid_size,local_coords);
    int globalIdx = globalIdx3d.x * ng * ng + globalIdx3d.y * ng + globalIdx3d.z;

    float4 this_out;
    this_out.x = x.x;
    this_out.y = y.x;
    this_out.z = z.x;
    this_out.w = globalIdx;

    int3 rhoIdx3d = HACCGPM::parallel::get_local_index(idx,local_grid_size.x,local_grid_size.y,local_grid_size.z);
    rhoIdx3d.x += overload;
    rhoIdx3d.y += overload;
    rhoIdx3d.z += overload;

    int3 overload_dims = make_int3(local_grid_size.x + 2*overload, local_grid_size.y + 2*overload, local_grid_size.z + 2*overload);

    int rhoIdx = rhoIdx3d.x * overload_dims.y * overload_dims.z + rhoIdx3d.y * overload_dims.z + rhoIdx3d.z;

    out[rhoIdx] = this_out;
}

CPUTimer_t launch_combine(float4* d_grad, deviceFFT_t* d_x, deviceFFT_t* d_y, deviceFFT_t* d_z, int numBlocks, int blockSize, int calls){
    getIndent(calls);
    return InvokeGPUKernel(combine,numBlocks,blockSize,d_grad,d_x,d_y,d_z);
}

CPUTimer_t launch_combine(float4* d_grad, deviceFFT_t* d_x, deviceFFT_t* d_y, deviceFFT_t* d_z, int3 local_grid_size, int3 local_coords, int overload, int nlocal, int ng, int world_rank, int numBlocks, int blockSize, int calls){
    getIndent(calls);
    return InvokeGPUKernelParallel(combine_parallel,numBlocks,blockSize,d_grad,d_x,d_y,d_z,local_grid_size,local_coords,overload,nlocal,ng);
}