#include "hip/hip_runtime.h"
#include "ic_kernels.hpp"

__global__ void transformDensityField(const deviceFFT_t* __restrict oldGrid, deviceFFT_t* __restrict outSx, deviceFFT_t* __restrict outSy, deviceFFT_t* __restrict outSz, double delta, double rl, double a, int ng){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    double d = (2*M_PI)/rl;

    int3 idx3d = HACCGPM::serial::get_index(idx,ng);
    float3 kmodes = HACCGPM::get_kmodes(idx3d,ng,d);

    double k2 = kmodes.x * kmodes.x + kmodes.y * kmodes.y + kmodes.z * kmodes.z;

    double k2mul = (1/k2);
    if (k2 == 0){
        k2mul = 0;
    }

    double mul = (1/delta) * k2mul;

    deviceFFT_t current = __ldg(&oldGrid[idx]);
    current.x *= mul;
    current.y *= mul;

    deviceFFT_t sx,sy,sz;

    sx.x = current.y * kmodes.x;
    sx.y = -current.x * kmodes.x;

    sy.x = current.y * kmodes.y;
    sy.y = -current.x * kmodes.y;

    sz.x = current.y * kmodes.z;
    sz.y = -current.x * kmodes.z;

    outSx[idx] = sx;
    outSy[idx] = sy;
    outSz[idx] = sz;

}

__global__ void transformDensityField(const deviceFFT_t* __restrict oldGrid, deviceFFT_t* __restrict outSx, deviceFFT_t* __restrict outSy, deviceFFT_t* __restrict outSz, double delta, double rl, double a, int ng, int nlocal, int world_rank, int3 local_grid_size, int3 local_coords, int3 dims){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx >= nlocal)return;

    double d = (2*M_PI)/rl;

    int3 idx3d = HACCGPM::parallel::get_global_index(idx,ng,local_grid_size,local_coords);
    float3 kmodes = HACCGPM::get_kmodes(idx3d,ng,d);

    double k2 = kmodes.x * kmodes.x + kmodes.y * kmodes.y + kmodes.z * kmodes.z;

    double k2mul = (1/k2);
    if (k2 == 0){
        k2mul = 0;
    }

    double mul = (1/delta) * k2mul;

    deviceFFT_t current = __ldg(&oldGrid[idx]);
    current.x *= mul;
    current.y *= mul;

    deviceFFT_t sx,sy,sz;

    sx.x = current.y * kmodes.x;
    sx.y = -current.x * kmodes.x;

    sy.x = current.y * kmodes.y;
    sy.y = -current.x * kmodes.y;

    sz.x = current.y * kmodes.z;
    sz.y = -current.x * kmodes.z;

    outSx[idx] = sx;
    outSy[idx] = sy;
    outSz[idx] = sz;

}