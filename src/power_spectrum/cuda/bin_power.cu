#include "hip/hip_runtime.h"
#include "power_kernels.hpp"

template<class T1, class T2>
__forceinline__ __device__ void calc_power_bins(T1* __restrict d_binVals, int* __restrict d_binCounts, T2 this_val, int idx, int3 idx3d, double minK, double binDelta, double rl, int ng){
    if ((idx3d.x == 0) && (idx3d.y == 0) && (idx3d.z == 0))return;
    if ((idx3d.x == ng/2) && (idx3d.y == ng/2) && (idx3d.z == ng/2))return;

    double d = (2*M_PI)/(rl);

    float3 kmodes = HACCGPM::get_kmodes(idx3d,ng,d);

    float kbin = sqrtf(kmodes.x*kmodes.x + kmodes.y*kmodes.y + kmodes.z*kmodes.z) - minK;
    int indx = (int)(kbin/binDelta);

    atomicAdd(&d_binVals[indx],this_val);
    atomicAdd(&d_binCounts[indx],1);
}

template<class T1, class T2>
__global__ void BinPower(const T1* __restrict d_grid, T2* __restrict d_binVals, int* __restrict d_binCounts, double minK, double binDelta, double rl, int ng){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    int3 idx3d = HACCGPM::serial::get_index(idx,ng);

    T1 this_val = __ldg(&d_grid[idx]);

    calc_power_bins(d_binVals,d_binCounts,this_val.x,idx,idx3d,minK,binDelta,rl,ng);

}

template __global__ void BinPower<deviceFFT_t,double>(const deviceFFT_t* __restrict,double* __restrict,int* __restrict, double, double, double, int);
template __global__ void BinPower<deviceFFT_t,float>(const deviceFFT_t* __restrict,float* __restrict,int* __restrict, double, double, double, int);
template __global__ void BinPower<floatFFT_t,double>(const floatFFT_t* __restrict,double* __restrict,int* __restrict, double, double, double, int);
template __global__ void BinPower<floatFFT_t,float>(const floatFFT_t* __restrict,float* __restrict,int* __restrict, double, double, double, int);

template<class T1, class T2>
__global__ void BinPower(const T1* __restrict d_grid, T2* __restrict d_binVals, int* __restrict d_binCounts, double minK, double binDelta, double rl, int ng, int nlocal, int world_rank, int3 local_grid_size, int3 local_coords, int3 dims){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= nlocal)return;

    int3 idx3d = HACCGPM::parallel::get_global_index(idx,ng,local_grid_size,local_coords);

    T1 this_val = __ldg(&d_grid[idx]);

    calc_power_bins(d_binVals,d_binCounts,this_val.x,idx,idx3d,minK,binDelta,rl,ng);

}

template __global__ void BinPower<deviceFFT_t,double>(const deviceFFT_t* __restrict,double* __restrict,int* __restrict, double, double, double, int,int,int,int3,int3,int3);
template __global__ void BinPower<deviceFFT_t,float>(const deviceFFT_t* __restrict,float* __restrict,int* __restrict, double, double, double, int,int,int,int3,int3,int3);
template __global__ void BinPower<floatFFT_t,double>(const floatFFT_t* __restrict,double* __restrict,int* __restrict, double, double, double, int,int,int,int3,int3,int3);
template __global__ void BinPower<floatFFT_t,float>(const floatFFT_t* __restrict,float* __restrict,int* __restrict, double, double, double, int,int,int,int3,int3,int3);