#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "haccgpm.hpp"

#define VerboseSwap


__global__ void swap_set_invalid(float4* __restrict d_pos, int mem_frac){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    if (idx >= mem_frac) return;
    d_pos[idx] = make_float4(0,0,0,-10);
}

__global__ void copy(float4* __restrict dest1, const float4* __restrict source1, float4* __restrict dest2, const float4* __restrict source2, int n){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    if (idx >= n)return;
    float4 my_particle = __ldg(&source1[idx]);
    float4 my_vel = __ldg(&source2[idx]);
    dest1[idx] = my_particle;
    dest2[idx] = my_vel;
}

__global__ void init_count_buffer(int* ns, const float4* __restrict d_pos, int n_particles, int ng, int3 local_grid_size, int3 local_coords, int3 dims){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    if (idx >= n_particles)return;
    float4 my_particle = __ldg(&d_pos[idx]);
    if (my_particle.w < -1)return;

    my_particle.x += (float)(local_grid_size.x * local_coords.x);
    my_particle.y += (float)(local_grid_size.y * local_coords.y);
    my_particle.z += (float)(local_grid_size.z * local_coords.z);

    my_particle.x += (float)ng;
    my_particle.y += (float)ng;
    my_particle.z += (float)ng;

    my_particle.x = fmod(my_particle.x,(float)ng);
    my_particle.y = fmod(my_particle.y,(float)ng);
    my_particle.z = fmod(my_particle.z,(float)ng);

    int3 dest_coords = make_int3(my_particle.x / local_grid_size.x,my_particle.y / local_grid_size.y,my_particle.z / local_grid_size.z);

    int dest_rank = dest_coords.x * dims.y * dims.z + dest_coords.y * dims.z + dest_coords.z;

    atomicAdd(&ns[dest_rank],1);
}

__global__ void init_load_buffer(float4* d_swap, int* ns, int* counts, float4* __restrict d_pos, float4* __restrict d_vel, int n_particles, int ng, int3 local_grid_size, int3 local_coords, int3 dims, int world_rank){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    if (idx >= n_particles)return;
    float4 my_particle = __ldg(&d_pos[idx]);
    if (my_particle.w < -1)return;

    my_particle.x += (float)(local_grid_size.x * local_coords.x);
    my_particle.y += (float)(local_grid_size.y * local_coords.y);
    my_particle.z += (float)(local_grid_size.z * local_coords.z);

    my_particle.x += (float)ng;
    my_particle.y += (float)ng;
    my_particle.z += (float)ng;

    my_particle.x = fmod(my_particle.x,(float)ng);
    my_particle.y = fmod(my_particle.y,(float)ng);
    my_particle.z = fmod(my_particle.z,(float)ng);

    int3 dest_coords = make_int3(my_particle.x / local_grid_size.x,my_particle.y / local_grid_size.y,my_particle.z / local_grid_size.z);

    int dest_rank = dest_coords.x * dims.y * dims.z + dest_coords.y * dims.z + dest_coords.z;

    if (dest_rank == world_rank)return;

    float3 grid_start = make_float3(dest_coords.x * local_grid_size.x, dest_coords.y * local_grid_size.y, dest_coords.z * local_grid_size.z);

    my_particle.x -= grid_start.x;
    my_particle.y -= grid_start.y;
    my_particle.z -= grid_start.z;

    int start = ns[dest_rank];

    int count = atomicAdd(&counts[dest_rank],1);

    int indx = start + count;

    d_swap[indx*2] = my_particle;
    d_swap[(indx*2)+1] = __ldg(&d_vel[idx]);

    d_pos[idx] = make_float4(0,0,0,-10);
    d_vel[idx] = make_float4(0,0,0,-10);

}

__global__ void findDuplicates(const float4* __restrict d_pos, float4* __restrict new_particles, int n_new, int n_particles){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    if (idx >= n_particles)return;
    float4 my_particle = __ldg(&d_pos[idx]);
    for (int i = 0; i < n_new; i++){
        float4 other_particle = __ldg(&new_particles[i]);
        if (my_particle.w == other_particle.w){
            //printf("DUPLICATE!!!\n");
            new_particles[i] = make_float4(0,0,0,-10);
            return;
        }
    }
}

__global__ void combineParticles(float4* __restrict d_pos, float4* __restrict d_vel, const float4* __restrict d_swap, int* d_count, int n_new, int n_particles){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    if (idx >= n_particles)return;
    float4 my_particle = __ldg(&d_pos[idx]);
    if (!(my_particle.w < -1))return;
    
    int my_idx = atomicAdd(d_count,1);
    if (my_idx >= n_new)return;
    float4 new_pos = __ldg(&d_swap[my_idx*2]);
    float4 new_vel = __ldg(&d_swap[(my_idx*2) + 1]);
    d_pos[idx] = new_pos;
    d_vel[idx] = new_vel;
}

CPUTimer_t HACCGPM::parallel::insertParticles(float4* d_pos, float4* d_vel, float4* h_swap, int n_new, int n_particles, int blockSize, int world_rank, int calls){
    int numBlocks = (n_particles + blockSize - 1) / blockSize;
    getIndent(calls);

    #ifdef VerboseSwap
    if(world_rank == 0)printf("%sinsertParticles was called with\n%s   blockSize %d\n%s   numBlocks %d\n",indent,indent,blockSize,indent,numBlocks);
    if(world_rank == 0)printf("%s   Copying to device\n",indent);
    #endif

    CPUTimer_t start = CPUTimer();

    float4* d_swap; cudaCall(hipMalloc,&d_swap,sizeof(float4)*2*n_new);
    cudaCall(hipMemcpy,d_swap,h_swap,sizeof(float4)*2*n_new,hipMemcpyHostToDevice);

    int* d_count; cudaCall(hipMalloc,&d_count,sizeof(int)*1);
    cudaCall(hipMemset,d_count,0,sizeof(int)*1);

    #ifdef VerboseSwap
    if(world_rank == 0)printf("%s      Copied to device\n",indent);
    #endif
    CPUTimer_t gpu_time = 0;
    gpu_time += InvokeGPUKernelParallel(combineParticles,numBlocks,blockSize,d_pos,d_vel,d_swap,d_count,n_new,n_particles);

    cudaCall(hipFree,d_swap);
    cudaCall(hipFree,d_count);

    CPUTimer_t end = CPUTimer();
    CPUTimer_t total_time = end-start;

    if(world_rank == 0)printf("%s   insertParticles took %llu us\n",indent,total_time);
    return gpu_time;
}

CPUTimer_t HACCGPM::parallel::LoadIntoBuffers(float4* h_swap, int* n_swaps, int* h_starts, float4* d_pos, float4* d_vel, int nlocal, int3 local_grid_size, int3 local_coords, int3 dims, int n_particles, int ng, int blockSize, int world_rank, int world_size, int calls){
    int numBlocks = (n_particles + blockSize - 1) / blockSize;

    getIndent(calls);

    #ifdef VerboseSwap
    if(world_rank == 0)printf("%sLoadIntoBuffers was called with\n%s   blockSize %d\n%s   numBlocks %d\n",indent,indent,blockSize,indent,numBlocks);
    #endif

    CPUTimer_t start = CPUTimer();
    CPUTimer_t gpu_time = 0;

    int* d_ns; cudaCall(hipMalloc,&d_ns,sizeof(int)*world_size);
    int* d_count; cudaCall(hipMalloc,&d_count,sizeof(int)*world_size);
    cudaCall(hipMemset,d_ns,0,sizeof(int) * world_size);
    cudaCall(hipMemset,d_count,0,sizeof(int) * world_size);
    gpu_time += InvokeGPUKernelParallel(init_count_buffer,numBlocks,blockSize,d_ns,d_pos,n_particles,ng,local_grid_size,local_coords,dims);
    cudaCall(hipMemcpy, n_swaps, d_ns, sizeof(int)*world_size, hipMemcpyDeviceToHost);
    
    h_starts[0] = 0;
    int transfer_size = 0;
    for (int i = 0; i < world_size; i++){
        if (i != world_rank){
            transfer_size += n_swaps[i];
        }
        if (i != (world_size - 1))h_starts[i+1] = transfer_size;
    }
    int* d_starts; cudaCall(hipMalloc,&d_starts,sizeof(int)*world_size);

    cudaCall(hipMemcpy,d_starts,h_starts,sizeof(int)*world_size,hipMemcpyHostToDevice);

    float4* d_swap; cudaCall(hipMalloc,&d_swap,sizeof(float4)*2*transfer_size);

    gpu_time += InvokeGPUKernelParallel(init_load_buffer,numBlocks,blockSize,d_swap,d_starts,d_count,d_pos,d_vel,n_particles,ng,local_grid_size,local_coords,dims,world_rank);

    cudaCall(hipMemcpy, h_swap, d_swap, sizeof(float4)*2*transfer_size, hipMemcpyDeviceToHost);

    cudaCall(hipFree,d_ns);
    cudaCall(hipFree,d_count);
    cudaCall(hipFree,d_swap);
    cudaCall(hipFree,d_starts);

    CPUTimer_t end = CPUTimer();
    CPUTimer_t total_time = end-start;

    if(world_rank == 0)printf("%s   LoadIntoBuffers took %llu us\n",indent,total_time);
    return gpu_time;
}