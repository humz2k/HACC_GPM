#include "hip/hip_runtime.h"
#include "../ic_kernels.hpp"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void initRNG(hiprandState *state, int seed){

  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(seed, idx, 0, &state[idx]);

}

__global__ void initRNG(hiprandState *state, int seed, int nlocal, int ng, int3 local_grid_size, int3 local_coords){

  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  if (idx >= nlocal)return;
  int3 idx3d = HACCGPM::parallel::get_global_index(idx,ng,local_grid_size,local_coords);
  int global_idx = idx3d.x * ng * ng + idx3d.y * ng + idx3d.z;
  hiprand_init(seed, global_idx, 0, &state[idx]);

}

template<class T>
__global__ void GenerateRealRandom(hiprandState* state, T* __restrict grid, int nlocal){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    if (idx >= nlocal)return;
    hostFFT_t amp = hiprand_normal_double(state + idx);
    T out;
    out.x = amp;
    out.y = 0;
    grid[idx] = out;
}

void launch_generate_rng(deviceFFT_t* d_grid1, int ng, int seed, int numBlocks, int blockSize, int calls){
    getIndent(calls);
    hiprandState* rngState; cudaCall(hipMalloc,&rngState,sizeof(hiprandState)*ng*ng*ng);
    InvokeGPUKernel(initRNG,numBlocks,blockSize,rngState,seed);
    InvokeGPUKernel(GenerateRealRandom,numBlocks,blockSize,rngState,d_grid1,ng*ng*ng);
    cudaCall(hipFree,rngState);
}

void launch_generate_rng(floatFFT_t* d_grid1, int ng, int seed, int numBlocks, int blockSize, int calls){
    getIndent(calls);
    hiprandState* rngState; cudaCall(hipMalloc,&rngState,sizeof(hiprandState)*ng*ng*ng);
    InvokeGPUKernel(initRNG,numBlocks,blockSize,rngState,seed);
    InvokeGPUKernel(GenerateRealRandom,numBlocks,blockSize,rngState,d_grid1,ng*ng*ng);
    cudaCall(hipFree,rngState);
}

void launch_generate_rng(deviceFFT_t* d_grid1, int ng, int seed, int nlocal, int3 local_grid_size, int3 local_coords, int world_rank, int numBlocks, int blockSize, int calls){
    getIndent(calls);
    hiprandState* rngState; cudaCall(hipMalloc,&rngState,sizeof(hiprandState)*nlocal);
    InvokeGPUKernelParallel(initRNG,numBlocks,blockSize,rngState,seed,nlocal,ng,local_grid_size,local_coords);
    InvokeGPUKernelParallel(GenerateRealRandom,numBlocks,blockSize,rngState,d_grid1,nlocal);
    cudaCall(hipFree,rngState);
}