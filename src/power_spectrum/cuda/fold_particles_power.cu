#include "hip/hip_runtime.h"
#include "power_kernels.hpp"

template<class T>
__global__ void foldParticles(T* __restrict d_pos, double ng, int np){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= (np*np*np))return;
    T my_particle = __ldg(&d_pos[idx]);
    my_particle.x /= ng;
    if (my_particle.x >= 0.5){
        my_particle.x -= 0.5;
    }
    my_particle.x *= 2 * ng;

    my_particle.y /= ng;
    if (my_particle.y >= 0.5){
        my_particle.y -= 0.5;
    }
    my_particle.y *= 2 * ng;

    my_particle.z /= ng;
    if (my_particle.z >= 0.5){
        my_particle.z -= 0.5;
    }
    my_particle.z *= 2 * ng;

    d_pos[idx] = my_particle;
}

template __global__ void foldParticles<float4>(float4* __restrict,double,int);
template __global__ void foldParticles<float3>(float3* __restrict,double,int);

__global__ void foldParticles(float4* __restrict d_pos, double ng, int3 local_grid_size, int3 local_coords){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    float4 my_particle = __ldg(&d_pos[idx]);

    if (my_particle.w < -1)return;if (my_particle.w < -1)return;

    my_particle.x += local_grid_size.x * local_coords.x;
    my_particle.y += local_grid_size.y * local_coords.y;
    my_particle.z += local_grid_size.z * local_coords.z;
    
    my_particle.x /= ng;
    if (my_particle.x >= 0.5){
        my_particle.x -= 0.5;
    }
    my_particle.x *= 2 * ng;

    my_particle.y /= ng;
    if (my_particle.y >= 0.5){
        my_particle.y -= 0.5;
    }
    my_particle.y *= 2 * ng;

    my_particle.z /= ng;
    if (my_particle.z >= 0.5){
        my_particle.z -= 0.5;
    }
    my_particle.z *= 2 * ng;

    my_particle.x -= local_grid_size.x * local_coords.x;
    my_particle.y -= local_grid_size.y * local_coords.y;
    my_particle.z -= local_grid_size.z * local_coords.z;

    d_pos[idx] = my_particle;
}

template<class T>
__global__ void cpy(T* __restrict dest, const T* __restrict source, int n){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= n)return;
    dest[idx] = __ldg(&source[idx]);
}

template __global__ void cpy<float4>(float4* __restrict, const float4* __restrict, int);
template __global__ void cpy<float3>(float3* __restrict, const float3* __restrict, int);