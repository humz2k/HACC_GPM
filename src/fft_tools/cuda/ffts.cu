#include <stdio.h>
#include <stdlib.h>
#include "haccgpm.hpp"

//#define VerboseFFT

#define FFTCacheSize 10

#define UsePlanManager

CPUTimer_t FFT_FORWARD_TIME_IP = 0;
CPUTimer_t FFT_FORWARD_PLAN_TIME_IP = 0;
CPUTimer_t FFT_BACKWARD_TIME_IP = 0;
CPUTimer_t FFT_BACKWARD_PLAN_TIME_IP = 0;
CPUTimer_t FFT_FORWARD_TIME_OP = 0;
CPUTimer_t FFT_FORWARD_PLAN_TIME_OP = 0;
CPUTimer_t FFT_BACKWARD_TIME_OP = 0;
CPUTimer_t FFT_BACKWARD_PLAN_TIME_OP = 0;

int FFT_FORWARD_CALLS_IP = 0;
int FFT_FORWARD_CALLS_OP = 0;
int FFT_BACKWARD_CALLS_IP = 0;
int FFT_BACKWARD_CALLS_OP = 0;

CPUTimer_t SINGLE_FFT_FORWARD_TIME_IP = 0;
CPUTimer_t SINGLE_FFT_FORWARD_PLAN_TIME_IP = 0;
CPUTimer_t SINGLE_FFT_BACKWARD_TIME_IP = 0;
CPUTimer_t SINGLE_FFT_BACKWARD_PLAN_TIME_IP = 0;
CPUTimer_t SINGLE_FFT_FORWARD_TIME_OP = 0;
CPUTimer_t SINGLE_FFT_FORWARD_PLAN_TIME_OP = 0;
CPUTimer_t SINGLE_FFT_BACKWARD_TIME_OP = 0;
CPUTimer_t SINGLE_FFT_BACKWARD_PLAN_TIME_OP = 0;

int SINGLE_FFT_FORWARD_CALLS_IP = 0;
int SINGLE_FFT_FORWARD_CALLS_OP = 0;
int SINGLE_FFT_BACKWARD_CALLS_IP = 0;
int SINGLE_FFT_BACKWARD_CALLS_OP = 0;

void printCufftError(hipfftResult error){
    if (error == HIPFFT_INVALID_PLAN){
        printf("HIPFFT_INVALID_PLAN\n");
    }
    if (error == HIPFFT_ALLOC_FAILED){
        printf("HIPFFT_ALLOC_FAILED\n");
    }
    if (error == HIPFFT_INVALID_VALUE){
        printf("HIPFFT_INVALID_VALUE\n");
    }
    if (error == HIPFFT_INTERNAL_ERROR){
        printf("HIPFFT_INTERNAL_ERROR\n");
    }
    if (error == HIPFFT_SETUP_FAILED){
        printf("HIPFFT_SETUP_FAILED\n");
    }
    if (error == HIPFFT_INVALID_SIZE){
        printf("HIPFFT_INVALID_SIZE\n");
    }
}

class PlanManager{
    public:
        hipfftHandle plans[FFTCacheSize];
        int ngs[FFTCacheSize];
        bool is_single[FFTCacheSize];
        int used;
        PlanManager(){
            //printf("FFT PlanManager:\n");
            //printf("   Initialized\n");
            used = 0;
        }
        ~PlanManager(){
            if (used != 0){
                #ifdef VerboseFFT
                printf("FFT PlanManager:\n");
                #endif
                for (int i = 0; i < used; i++){
                    #ifdef VerboseFFT
                    printf("   Freeing plan %d\n",ngs[i]);
                    #endif
                    hipfftDestroy(plans[i]);
                }
            } else {
                //printf("   No plans to free\n");
            }
        }
        hipfftHandle get_plan(int ng, bool single_precision, int calls){
            getIndent(calls);
            #ifdef VerboseFFT
            printf("%sFFT PlanManager:\n",indent);
            printf("%s   Searching for cached plan %d\n",indent,ng);
            #endif
            for (int i = 0; i < used; i++){
                if ((ngs[i] == ng) && (is_single[i] == single_precision)){
                    #ifdef VerboseFFT
                    printf("%s   Found cached plan %d\n",indent,ngs[i]);
                    #endif
                    return plans[i];
                }
            }
            if (used >= FFTCacheSize){
                #ifdef VerboseFFT
                printf("%s   No more space to create new plan!!!\n",indent);
                #endif
                exit(1);
            }
            #ifdef VerboseFFT
            printf("%s   No cached plan %d found, creating one\n",indent,ng);
            #endif
            if (single_precision){
                hipfftResult result = hipfftPlan3d(&plans[used], ng, ng, ng, HIPFFT_C2C);
                if (result != HIPFFT_SUCCESS){
                    printf("CUFFT error: Plan creation failed with ");
                    printCufftError(result);
                    exit(1);
                };

            } else{
                hipfftResult result = hipfftPlan3d(&plans[used], ng, ng, ng, HIPFFT_Z2Z);
                if (result != HIPFFT_SUCCESS){
                    printf("CUFFT error: Plan creation failed with ");
                    printCufftError(result);
                    exit(1);
                };
            }
            ngs[used] = ng;
            is_single[used] = single_precision;
            used++;
            return plans[used-1];
        }
};

#ifdef UsePlanManager
PlanManager plan_manager;
#endif

void HACCGPM::serial::fft_cache_plan(int ng){
    #ifdef UsePlanManager
    plan_manager.get_plan(ng,false,0);
    #endif
}

void HACCGPM::serial::forward_fft(deviceFFT_t* data, deviceFFT_t* out, int ng, int calls){

    CPUTimer_t start = CPUTimer();

    getIndent(calls);

    #ifdef VerboseFFT
    printf("%sforward_fft (out of place) was called with\n%s   ng %d\n",indent,indent,ng);
    #endif

    hipfftHandle plan;
    #ifdef UsePlanManager
    plan = plan_manager.get_plan(ng,false,calls+1);
    #else
    printf("%s   Creating plan %d\n",indent,ng);
    if (hipfftPlan3d(&plan, ng, ng, ng, HIPFFT_Z2Z) != HIPFFT_SUCCESS){
        printf("CUFFT error: Plan creation failed\n");
        return;
    };
    #endif

    CPUTimer_t plan_t = CPUTimer();
    #ifdef VerboseFFT
    printf("%s   Executing Z2Z HIPFFT_FORWARD\n",indent);
    #endif
    if (hipfftExecZ2Z(plan, data, out, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
        printf("CUFFT error: ExecZ2Z Forward failed\n");
        return;	
    }
    hipDeviceSynchronize();
    #ifndef UsePlanManager
    #ifdef VerboseFFT
    printf("%s   Destroying plan %d\n",indent,ng);
    #endif
    hipfftDestroy(plan);
    #endif

    CPUTimer_t end = CPUTimer();
    CPUTimer_t plan_time = plan_t - start;
    CPUTimer_t t = end-start;
    #ifdef VerboseFFT
    printf("%s   forward_fft (out of place) took %llu us (%llu us planning)\n",indent,t,plan_time);
    #else
    printf("forward_fft (out of place, double): %llu us (%llu us planning)\n",t,plan_time);
    #endif
    FFT_FORWARD_CALLS_OP++;
    FFT_FORWARD_TIME_OP += t;
    FFT_FORWARD_PLAN_TIME_OP += plan_time;
}

void HACCGPM::serial::forward_fft(deviceFFT_t* data, int ng, int calls){

    CPUTimer_t start = CPUTimer();

    getIndent(calls);

    #ifdef VerboseFFT
    printf("%sforward_fft (in place) was called with\n%s   ng %d\n",indent,indent,ng);
    #endif

    hipfftHandle plan;
    #ifdef UsePlanManager
    plan = plan_manager.get_plan(ng,false,calls+1);
    #else
    #ifdef VerboseFFT
    printf("%s   Creating plan %d\n",indent,ng);
    #endif
    if (hipfftPlan3d(&plan, ng, ng, ng, HIPFFT_Z2Z) != HIPFFT_SUCCESS){
        printf("CUFFT error: Plan creation failed\n");
        return;
    };
    #endif
    
    CPUTimer_t plan_t = CPUTimer();
    #ifdef VerboseFFT
    printf("%s   Executing Z2Z HIPFFT_FORWARD\n",indent);
    #endif
    if (hipfftExecZ2Z(plan, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
        printf("CUFFT error: ExecZ2Z Forward failed\n");
        return;	
    }

    hipDeviceSynchronize();

    #ifndef UsePlanManager
    printf("%s   Destroying plan %d\n",indent,ng);
    hipfftDestroy(plan);
    #endif

    CPUTimer_t end = CPUTimer();
    CPUTimer_t plan_time = plan_t - start;
    CPUTimer_t t = end-start;
    #ifdef VerboseFFT
    printf("%s   forward_fft (in place) took %llu us (%llu us planning)\n",indent,t,plan_time);
    #else
    printf("forward_fft (in place, double): %llu us (%llu us planning)\n",t,plan_time);
    #endif
    FFT_FORWARD_CALLS_IP++;
    FFT_FORWARD_TIME_IP += t;
    FFT_FORWARD_PLAN_TIME_IP += plan_time;
}

void HACCGPM::serial::forward_fft(floatFFT_t* data, int ng, int calls){

    CPUTimer_t start = CPUTimer();

    getIndent(calls);

    #ifdef VerboseFFT
    printf("%sforward_fft (in place) was called with\n%s   ng %d\n",indent,indent,ng);
    #endif

    hipfftHandle plan;
    #ifdef UsePlanManager
    plan = plan_manager.get_plan(ng,true,calls+1);
    #else
    #ifdef VerboseFFT
    printf("%s   Creating plan %d\n",indent,ng);
    #endif
    if (hipfftPlan3d(&plan, ng, ng, ng, HIPFFT_C2C) != HIPFFT_SUCCESS){
        printf("CUFFT error: Plan creation failed\n");
        return;
    };
    #endif
    
    CPUTimer_t plan_t = CPUTimer();
    #ifdef VerboseFFT
    printf("%s   Executing C2C HIPFFT_FORWARD\n",indent);
    #endif
    if (hipfftExecC2C(plan, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
        printf("CUFFT error: ExecC2C Forward failed\n");
        return;	
    }

    hipDeviceSynchronize();

    #ifndef UsePlanManager
    printf("%s   Destroying plan %d\n",indent,ng);
    hipfftDestroy(plan);
    #endif

    CPUTimer_t end = CPUTimer();
    CPUTimer_t plan_time = plan_t - start;
    CPUTimer_t t = end-start;
    #ifdef VerboseFFT
    printf("%s   forward_fft (in place) took %llu us (%llu us planning)\n",indent,t,plan_time);
    #else
    printf("forward_fft (in place, float): %llu us (%llu us planning)\n",t,plan_time);
    #endif
    SINGLE_FFT_FORWARD_CALLS_IP++;
    SINGLE_FFT_FORWARD_TIME_IP += t;
    SINGLE_FFT_FORWARD_PLAN_TIME_IP += plan_time;
}

void HACCGPM::serial::backward_fft(deviceFFT_t* data, deviceFFT_t* out, int ng, int calls){

    CPUTimer_t start = CPUTimer();

    getIndent(calls);

    #ifdef VerboseFFT
    printf("%sbackward_fft (out of place) was called with\n%s   ng %d\n",indent,indent,ng);
    #endif

    hipfftHandle plan;
    #ifdef UsePlanManager
    plan = plan_manager.get_plan(ng,false,calls+1);
    #else
    printf("%s   Creating plan %d\n",indent,ng);
    if (hipfftPlan3d(&plan, ng, ng, ng, HIPFFT_Z2Z) != HIPFFT_SUCCESS){
        printf("CUFFT error: Plan creation failed\n");
        return;
    };
    #endif

    CPUTimer_t plan_t = CPUTimer();
    #ifdef VerboseFFT
    printf("%s   Executing Z2Z HIPFFT_BACKWARD\n",indent);
    #endif
    if (hipfftExecZ2Z(plan, data, out, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
        printf("CUFFT error: ExecZ2Z Backward failed\n");
        return;	
    }
    hipDeviceSynchronize();

    #ifndef UsePlanManager
    #ifdef VerboseFFT
    printf("%s   Destroying plan %d\n",indent,ng);
    #endif
    hipfftDestroy(plan);
    #endif

    CPUTimer_t end = CPUTimer();
    CPUTimer_t plan_time = plan_t - start;
    CPUTimer_t t = end-start;
    #ifdef VerboseFFT
    printf("%s   inverse_fft (out of place) took %llu us (%llu us planning)\n",indent,t,plan_time);
    #else
    printf("inverse_fft (out of place, double) took %llu us (%llu us planning)\n",t,plan_time);
    #endif
    FFT_BACKWARD_CALLS_OP++;
    FFT_BACKWARD_TIME_OP += t;
    FFT_BACKWARD_PLAN_TIME_OP += plan_time;
}

void HACCGPM::serial::backward_fft(deviceFFT_t* data, int ng, int calls){

    CPUTimer_t start = CPUTimer();

    getIndent(calls);

    #ifdef VerboseFFT
    printf("%sbackward_fft (in place) was called with\n%s   ng %d\n",indent,indent,ng);
    #endif

    hipfftHandle plan;
    #ifdef UsePlanManager
    plan = plan_manager.get_plan(ng,false,calls+1);
    #else
    #ifdef VerboseFFT
    printf("%s   Creating plan %d\n",indent,ng);
    #endif
    if (hipfftPlan3d(&plan, ng, ng, ng, HIPFFT_Z2Z) != HIPFFT_SUCCESS){
        printf("CUFFT error: Plan creation failed\n");
        return;
    };
    #endif
    CPUTimer_t plan_t = CPUTimer();
    #ifdef VerboseFFT
    printf("%s   Executing Z2Z HIPFFT_BACKWARD\n",indent);
    #endif
    if (hipfftExecZ2Z(plan, data, data, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
        printf("CUFFT error: ExecZ2Z Backward failed\n");
        return;	
    }
    hipDeviceSynchronize();
    #ifndef UsePlanManager
    #ifdef VerboseFFT
    printf("%s   Destroying plan %d\n",indent,ng);
    #endif
    hipfftDestroy(plan);
    #endif
    CPUTimer_t end = CPUTimer();
    CPUTimer_t plan_time = plan_t - start;
    CPUTimer_t t = end-start;
    #ifdef VerboseFFT
    printf("%s   inverse_fft (in place) took %llu us (%llu us planning)\n",indent,t,plan_time);
    #else
    printf("inverse_fft (in place, double): %llu us (%llu us planning)\n",t,plan_time);
    #endif
    FFT_BACKWARD_CALLS_IP++;
    FFT_BACKWARD_TIME_IP += t;
    FFT_BACKWARD_PLAN_TIME_IP += plan_time;
}

void HACCGPM::serial::backward_fft(floatFFT_t* data, int ng, int calls){

    CPUTimer_t start = CPUTimer();

    getIndent(calls);

    #ifdef VerboseFFT
    printf("%sbackward_fft (in place) was called with\n%s   ng %d\n",indent,indent,ng);
    #endif

    hipfftHandle plan;
    #ifdef UsePlanManager
    plan = plan_manager.get_plan(ng,true,calls+1);
    #else
    #ifdef VerboseFFT
    printf("%s   Creating plan %d\n",indent,ng);
    #endif
    if (hipfftPlan3d(&plan, ng, ng, ng, HIPFFT_C2C) != HIPFFT_SUCCESS){
        printf("CUFFT error: Plan creation failed\n");
        return;
    };
    #endif
    CPUTimer_t plan_t = CPUTimer();
    #ifdef VerboseFFT
    printf("%s   Executing C2C HIPFFT_BACKWARD\n",indent);
    #endif
    if (hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
        printf("CUFFT error: ExecC2C Backward failed\n");
        return;	
    }
    hipDeviceSynchronize();
    #ifndef UsePlanManager
    #ifdef VerboseFFT
    printf("%s   Destroying plan %d\n",indent,ng);
    #endif
    hipfftDestroy(plan);
    #endif
    CPUTimer_t end = CPUTimer();
    CPUTimer_t plan_time = plan_t - start;
    CPUTimer_t t = end-start;
    #ifdef VerboseFFT
    printf("%s   inverse_fft (in place) took %llu us (%llu us planning)\n",indent,t,plan_time);
    #else
    printf("inverse_fft (in place, float): %llu us (%llu us planning)\n",t,plan_time);
    #endif
    SINGLE_FFT_BACKWARD_CALLS_IP++;
    SINGLE_FFT_BACKWARD_TIME_IP += t;
    SINGLE_FFT_BACKWARD_PLAN_TIME_IP += plan_time;
}

void HACCGPM::serial::printFFTTimes(){
    if (FFT_FORWARD_CALLS_OP != 0){
        printf("   forward_fft (d-op) -> calls: %10d | total: %10llu us | cpu: %10llu us | gpu: %10llu us | mean: %10.2f us\n",FFT_FORWARD_CALLS_OP,FFT_FORWARD_TIME_OP,FFT_FORWARD_PLAN_TIME_OP,FFT_FORWARD_TIME_OP - FFT_FORWARD_PLAN_TIME_OP,((float)FFT_FORWARD_TIME_OP)/((float)(FFT_FORWARD_CALLS_OP)));
    }
    if (FFT_FORWARD_CALLS_IP != 0){
        printf("   forward_fft (d-ip) -> calls: %10d | total: %10llu us | cpu: %10llu us | gpu: %10llu us | mean: %10.2f us\n",FFT_FORWARD_CALLS_IP,FFT_FORWARD_TIME_IP,FFT_FORWARD_PLAN_TIME_IP,FFT_FORWARD_TIME_IP - FFT_FORWARD_PLAN_TIME_IP,((float)FFT_FORWARD_TIME_IP)/((float)(FFT_FORWARD_CALLS_IP)));
    }
    if (FFT_BACKWARD_CALLS_OP != 0){
        printf("   inverse_fft (d-op) -> calls: %10d | total: %10llu us | cpu: %10llu us | gpu: %10llu us | mean: %10.2f us\n",FFT_BACKWARD_CALLS_OP,FFT_BACKWARD_TIME_OP,FFT_BACKWARD_PLAN_TIME_OP,FFT_BACKWARD_TIME_OP - FFT_BACKWARD_PLAN_TIME_OP,((float)FFT_BACKWARD_TIME_OP)/((float)(FFT_BACKWARD_CALLS_OP)));
    }
    if (FFT_BACKWARD_CALLS_IP != 0){
        printf("   inverse_fft (d-ip) -> calls: %10d | total: %10llu us | cpu: %10llu us | gpu: %10llu us | mean: %10.2f us\n",FFT_BACKWARD_CALLS_IP,FFT_BACKWARD_TIME_IP,FFT_BACKWARD_PLAN_TIME_IP,FFT_BACKWARD_TIME_IP - FFT_BACKWARD_PLAN_TIME_IP,((float)FFT_BACKWARD_TIME_IP)/((float)(FFT_BACKWARD_CALLS_IP)));
    }

    if (SINGLE_FFT_FORWARD_CALLS_OP != 0){
        printf("   forward_fft (f-op) -> calls: %10d | total: %10llu us | cpu: %10llu us | gpu: %10llu us | mean: %10.2f us\n",SINGLE_FFT_FORWARD_CALLS_OP,SINGLE_FFT_FORWARD_TIME_OP,SINGLE_FFT_FORWARD_PLAN_TIME_OP,SINGLE_FFT_FORWARD_TIME_OP - SINGLE_FFT_FORWARD_PLAN_TIME_OP,((float)SINGLE_FFT_FORWARD_TIME_OP)/((float)(SINGLE_FFT_FORWARD_CALLS_OP)));
    }
    if (SINGLE_FFT_FORWARD_CALLS_IP != 0){
        printf("   forward_fft (f-ip) -> calls: %10d | total: %10llu us | cpu: %10llu us | gpu: %10llu us | mean: %10.2f us\n",SINGLE_FFT_FORWARD_CALLS_IP,SINGLE_FFT_FORWARD_TIME_IP,SINGLE_FFT_FORWARD_PLAN_TIME_IP,SINGLE_FFT_FORWARD_TIME_IP - SINGLE_FFT_FORWARD_PLAN_TIME_IP,((float)SINGLE_FFT_FORWARD_TIME_IP)/((float)(SINGLE_FFT_FORWARD_CALLS_IP)));
    }
    if (SINGLE_FFT_BACKWARD_CALLS_OP != 0){
        printf("   inverse_fft (f-op) -> calls: %10d | total: %10llu us | cpu: %10llu us | gpu: %10llu us | mean: %10.2f us\n",SINGLE_FFT_BACKWARD_CALLS_OP,SINGLE_FFT_BACKWARD_TIME_OP,SINGLE_FFT_BACKWARD_PLAN_TIME_OP,SINGLE_FFT_BACKWARD_TIME_OP - SINGLE_FFT_BACKWARD_PLAN_TIME_OP,((float)SINGLE_FFT_BACKWARD_TIME_OP)/((float)(SINGLE_FFT_BACKWARD_CALLS_OP)));
    }
    if (SINGLE_FFT_BACKWARD_CALLS_IP != 0){
        printf("   inverse_fft (f-ip) -> calls: %10d | total: %10llu us | cpu: %10llu us | gpu: %10llu us | mean: %10.2f us\n",SINGLE_FFT_BACKWARD_CALLS_IP,SINGLE_FFT_BACKWARD_TIME_IP,SINGLE_FFT_BACKWARD_PLAN_TIME_IP,SINGLE_FFT_BACKWARD_TIME_IP - SINGLE_FFT_BACKWARD_PLAN_TIME_IP,((float)SINGLE_FFT_BACKWARD_TIME_IP)/((float)(SINGLE_FFT_BACKWARD_CALLS_IP)));
    }
}