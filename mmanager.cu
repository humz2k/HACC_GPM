#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "stdio.h"
#include "haccgpm.hpp"

HACCGPM::parallel::MemoryManager::MemoryManager(HACCGPM::Params params){
    world_rank = params.world_rank;
    if (params.world_rank == 0)printf("MemoryManager:\n   Allocating d_vel,d_pos,d_greens,d_grid1,d_grid2,d_grad...\n");
    int mem_frac = params.nlocal * params.frac;
    if (params.world_rank == 0)printf("   Mem frac: %d / %d\n",mem_frac,params.nlocal);
    cudaCall(hipMalloc,&d_vel,sizeof(float4)*mem_frac);
    if (params.world_rank == 0)printf("   Allocated d_vel: %lu bytes.\n",sizeof(float4)*mem_frac);

    cudaCall(hipMalloc,&d_pos,sizeof(float4)*mem_frac);
    if (params.world_rank == 0)printf("   Allocated d_pos: %lu bytes.\n",sizeof(float4)*mem_frac);

    cudaCall(hipMalloc,&d_greens,sizeof(hostFFT_t)*params.nlocal);
    if (params.world_rank == 0)printf("   Allocated d_greens: %lu bytes.\n",sizeof(hostFFT_t)*params.nlocal);

    cudaCall(hipMalloc,&d_grid1,sizeof(deviceFFT_t)*params.nlocal);
    if (params.world_rank == 0)printf("   Allocated d_grid: %lu bytes.\n",sizeof(deviceFFT_t)*params.nlocal);

    cudaCall(hipMalloc,&d_grid2,sizeof(deviceFFT_t)*params.nlocal);
    if (params.world_rank == 0)printf("   Allocated d_grid: %lu bytes.\n",sizeof(deviceFFT_t)*params.nlocal);

    cudaCall(hipMalloc,&d_grad,sizeof(float4)*mem_frac);
    if (params.world_rank == 0)printf("   Allocated d_grad: %lu bytes.\n",sizeof(float4)*mem_frac);
}

HACCGPM::parallel::MemoryManager::~MemoryManager(){
    if (world_rank == 0)printf("MemoryManager:\n   Freeing d_vel,d_pos,d_greens,d_grid1,d_grid2,d_grad...\n");
    
    cudaCall(hipFree,d_pos);
    cudaCall(hipFree,d_vel);
    cudaCall(hipFree,d_greens);
    cudaCall(hipFree,d_grid1);
    cudaCall(hipFree,d_grid2);
    cudaCall(hipFree,d_grad);

    if (world_rank == 0)printf("      Freed d_vel,d_pos,d_greens,d_grid1,d_grid2,d_grad.\n");
}

HACCGPM::serial::MemoryManager::MemoryManager(HACCGPM::Params params){
    printf("MemoryManager:\n   Allocating d_vel,d_pos,d_greens,d_grid,d_grad...\n");
    
    cudaCall(hipMalloc,&d_vel,sizeof(float4)*params.ng*params.ng*params.ng);
    printf("   Allocated d_vel: %lu bytes.\n",sizeof(float4)*params.ng*params.ng*params.ng);

    cudaCall(hipMalloc,&d_pos,sizeof(float4)*params.ng*params.ng*params.ng);
    printf("   Allocated d_pos: %lu bytes.\n",sizeof(float4)*params.ng*params.ng*params.ng);

    cudaCall(hipMalloc,&d_greens,sizeof(hostFFT_t)*params.ng*params.ng*params.ng);
    printf("   Allocated d_greens: %lu bytes.\n",sizeof(hostFFT_t)*params.ng*params.ng*params.ng);

    cudaCall(hipMalloc,&d_grid,sizeof(deviceFFT_t)*params.ng*params.ng*params.ng);
    printf("   Allocated d_grid: %lu bytes.\n",sizeof(deviceFFT_t)*params.ng*params.ng*params.ng);

    cudaCall(hipMalloc,&d_grad,sizeof(float4)*params.ng*params.ng*params.ng);
    printf("   Allocated d_grad: %lu bytes.\n",sizeof(float4)*params.ng*params.ng*params.ng);
}

HACCGPM::serial::MemoryManager::~MemoryManager(){
    printf("MemoryManager:\n   Freeing d_vel,d_pos,d_greens,d_grid,d_grad...\n");
    
    cudaCall(hipFree,d_pos);
    cudaCall(hipFree,d_vel);
    cudaCall(hipFree,d_greens);
    cudaCall(hipFree,d_grid);
    cudaCall(hipFree,d_grad);

    printf("      Freed d_vel,d_pos,d_greens,d_grid,d_grad.\n");
}