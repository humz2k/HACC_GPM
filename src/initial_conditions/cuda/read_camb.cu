#include "../ic_kernels.hpp"

#ifndef NOPYTHON
#include "../cambTools/ccamb.h"
#endif

CPUTimer_t launch_get_pk(hostFFT_t* d_pkScale, double z, const char* fname, int ng, double rl, int calls){

    CPUTimer_t start = CPUTimer();

    hostFFT_t* h_tmp = (hostFFT_t*)malloc(sizeof(hostFFT_t)*ng*ng*ng);

    get_pk(fname,h_tmp,z,ng,rl,calls+1);

    cudaCall(hipMemcpy, d_pkScale, h_tmp, sizeof(hostFFT_t)*ng*ng*ng, hipMemcpyHostToDevice);
    
    free(h_tmp);

    CPUTimer_t end = CPUTimer();

    return end - start;

}

CPUTimer_t launch_get_pk(hostFFT_t* d_pkScale, double z, const char* fname, int ng, double rl, int nlocal, int world_rank, int calls){

    CPUTimer_t start = CPUTimer();

    hostFFT_t* h_tmp = (hostFFT_t*)malloc(sizeof(hostFFT_t)*nlocal);

    get_pk_parallel(fname,h_tmp,z,ng,rl,nlocal,world_rank,calls+1);

    cudaCall(hipMemcpy, d_pkScale, h_tmp, sizeof(hostFFT_t)*nlocal, hipMemcpyHostToDevice);

    free(h_tmp);

    CPUTimer_t end = CPUTimer();

    return end - start;

}