#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "haccgpm.hpp"

#define VerboseUpdate

CPUTimer_t CIC_TIME = 0;
CPUTimer_t CIC_KERNEL_TIME = 0;
int CIC_CALLS = 0;

CPUTimer_t UPDATE_POS_TIME = 0;
CPUTimer_t UPDATE_POS_KERNEL_TIME = 0;
int UPDATE_POS_CALLS = 0;

CPUTimer_t UPDATE_VEL_TIME = 0;
CPUTimer_t UPDATE_VEL_KERNEL_TIME = 0;
int UPDATE_VEL_CALLS = 0;

__global__ void UpdatePosKernel(float4* __restrict d_pos, const float4* __restrict d_vel, float prefactor, float ng){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    float4 my_pos = __ldg(&d_pos[idx]);
    float4 my_vel = __ldg(&d_vel[idx]);
    my_pos.x += my_vel.x * prefactor;
    my_pos.y += my_vel.y * prefactor;
    my_pos.z += my_vel.z * prefactor;
    my_pos.x = fmod(my_pos.x + ng,ng);
    my_pos.y = fmod(my_pos.y + ng,ng);
    my_pos.z = fmod(my_pos.z + ng,ng);
    //if ((my_pos.x < 0 || my_pos.x >= ng) || (my_pos.y < 0 || my_pos.y >= ng) || (my_pos.z < 0 || my_pos.z >= ng)){
    //    printf("%g %g %g\n",my_pos.x,my_pos.y,my_pos.z);
    //    printf("FUCK!!!\n");
    //}
    d_pos[idx] = my_pos;
}

__global__ void ICICKernel(float4* __restrict d_vel, const float4* __restrict d_grad, const float4* __restrict my_pos, double deltaT, double fscal, int ng){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    float4 my_particle = __ldg(&my_pos[idx]);
    float3 my_deltaV = make_float3(0.0,0.0,0.0);
    int i = my_particle.x;
    int j = my_particle.y;
    int k = my_particle.z;

    float diffx = (my_particle.x - (float)i);
    float diffy = (my_particle.y - (float)j);
    float diffz = (my_particle.z - (float)k);

    for (int x = 0; x < 2; x++){
        for (int y = 0; y < 2; y++){
            for (int z = 0; z < 2; z++){
                int nx = (i + x)%ng;
                int ny = (j + y)%ng;
                int nz = (k + z)%ng;
                int indx = (nx)*ng*ng + (ny)*ng + nz;

                float dx = diffx;
                if (x == 0){
                    dx = 1 - dx;
                }
                float dy = diffy;
                if (y == 0){
                    dy = 1 - dy;
                }
                float dz = diffz;
                if (z == 0){
                    dz = 1 - dz;
                }

                float4 grad = __ldg(&d_grad[indx]);

                float mul = dx*dy*dz * deltaT * (fscal);//* (1.0f/((double)(ng*ng*ng)));// (1.0f/((double)(ng*ng*ng)));// * deltaT * fscal * (1.0f/((double)(ng*ng*ng)));
                my_deltaV.x += mul*grad.x;
                my_deltaV.y += mul*grad.y;
                my_deltaV.z += mul*grad.z;

                //atomicAdd(&grid[indx].x,(double)mul);
            }
        }
    }

    float4 my_vel = __ldg(&d_vel[idx]);
    my_vel.x += my_deltaV.x;
    my_vel.y += my_deltaV.y;
    my_vel.z += my_deltaV.z;

    d_vel[idx] = my_vel;

}

__global__ void CICKernel(deviceFFT_t* __restrict grid, const float4* __restrict my_pos, int ng, float mass){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    float4 my_particle = __ldg(&my_pos[idx]);
    int i = my_particle.x;
    int j = my_particle.y;
    int k = my_particle.z;

    float diffx = (my_particle.x - (float)i);
    float diffy = (my_particle.y - (float)j);
    float diffz = (my_particle.z - (float)k);

    for (int x = 0; x < 2; x++){
        for (int y = 0; y < 2; y++){
            for (int z = 0; z < 2; z++){
                int nx = (i + x)%ng;
                int ny = (j + y)%ng;
                int nz = (k + z)%ng;
                int indx = (nx)*ng*ng + (ny)*ng + nz;

                float dx = diffx;
                if (x == 0){
                    dx = 1 - dx;
                }
                float dy = diffy;
                if (y == 0){
                    dy = 1 - dy;
                }
                float dz = diffz;
                if (z == 0){
                    dz = 1 - dz;
                }

                float mul = dx*dy*dz*mass; //* (1.0f/(ng*ng*ng));

                atomicAdd(&grid[indx].x,(double)mul);
            }
        }
    }

}

__global__ void CICKernel(float* __restrict grid, const float4* __restrict my_pos, int ng, float mass){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    float4 my_particle = __ldg(&my_pos[idx]);
    int i = my_particle.x;
    int j = my_particle.y;
    int k = my_particle.z;

    float diffx = (my_particle.x - (float)i);
    float diffy = (my_particle.y - (float)j);
    float diffz = (my_particle.z - (float)k);

    for (int x = 0; x < 2; x++){
        for (int y = 0; y < 2; y++){
            for (int z = 0; z < 2; z++){
                int nx = (i + x)%ng;
                int ny = (j + y)%ng;
                int nz = (k + z)%ng;
                int indx = (nx)*ng*ng + (ny)*ng + nz;

                float dx = diffx;
                if (x == 0){
                    dx = 1 - dx;
                }
                float dy = diffy;
                if (y == 0){
                    dy = 1 - dy;
                }
                float dz = diffz;
                if (z == 0){
                    dz = 1 - dz;
                }

                float mul = dx*dy*dz*mass; //* (1.0f/(ng*ng*ng));

                atomicAdd(&grid[indx],mul);
            }
        }
    }
}

__global__ void float2complex(deviceFFT_t* __restrict d_out, const float* __restrict d_in, int n){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    if(idx >= n)return;
    float my_grid = __ldg(&d_in[idx]);
    deviceFFT_t out;
    out.x = my_grid;
    out.y = 0;
    d_out[idx] = out;
}

__global__ void CICKernelParallel(float* __restrict d_grid, const float4* __restrict d_pos, int ng, int3 local_grid_size, int n_particles, float mass){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    if (idx >= n_particles)return;

    float4 my_particle = __ldg(&d_pos[idx]);
    if (my_particle.w < -1)return;

    int i = my_particle.x;
    int j = my_particle.y;
    int k = my_particle.z;

    float diffx = (my_particle.x - (float)i);
    float diffy = (my_particle.y - (float)j);
    float diffz = (my_particle.z - (float)k);

    for (int x = 0; x < 2; x++){
        for (int y = 0; y < 2; y++){
            for (int z = 0; z < 2; z++){

                int nx = (i + x);
                int ny = (j + y);
                int nz = (k + z);
                if ((nx < 0) || (nx >= local_grid_size.x) || (ny < 0) || (ny >= local_grid_size.y) || (nz < 0) || (nz >= local_grid_size.z))continue;

                int indx = (nx)*(local_grid_size.y)*(local_grid_size.z) + (ny)*(local_grid_size.z) + nz;

                float dx = diffx;
                if (x == 0){
                    dx = 1 - dx;
                }
                float dy = diffy;
                if (y == 0){
                    dy = 1 - dy;
                }
                float dz = diffz;
                if (z == 0){
                    dz = 1 - dz;
                }

                float mul = dx*dy*dz*mass; //* (1.0f/(ng*ng*ng));

                atomicAdd(&d_grid[indx],mul);

            }
        }
    }
}

void HACCGPM::parallel::CIC(deviceFFT_t* d_grid, float* d_tempgrid, float4* d_pos, int ng, int n_particles, int* local_grid_size_, int blockSize, int world_rank, int world_size, int calls){
    CPUTimer_t start = CPUTimer();
    int numBlocks = (n_particles + (blockSize - 1))/blockSize;
    int3 local_grid_size = make_int3(local_grid_size_[0],local_grid_size_[1],local_grid_size_[2]);
    getIndent(calls);
    #ifdef VerboseUpdate
    if (world_rank == 0)printf("%sCIC was called with\n%s   blockSize %d\n%s   numBlocks %d\n",indent,indent,blockSize,indent,numBlocks);
    #endif
    cudaCall(hipMemset,d_tempgrid,0,sizeof(float)*(local_grid_size.x)*(local_grid_size.y)*(local_grid_size.z));
    CIC_KERNEL_TIME += InvokeGPUKernelParallel(CICKernelParallel,numBlocks,blockSize,d_tempgrid,d_pos,ng,local_grid_size,n_particles,1.0f);
    numBlocks = (local_grid_size.x*local_grid_size.y*local_grid_size.z + (blockSize - 1))/blockSize;
    InvokeGPUKernelParallel(float2complex,numBlocks,blockSize,d_grid,d_tempgrid,local_grid_size.x*local_grid_size.y*local_grid_size.z);

    //HACCGPM::parallel::gridExchange(d_extragrid,local_grid_size,world_rank,world_size,blockSize);
    
    CPUTimer_t end = CPUTimer();
    CPUTimer_t t = end-start;
    if (world_rank == 0)printf("%s   CIC took %llu us\n",indent,t);
    CIC_TIME += t;
    CIC_CALLS += 1;
}

void HACCGPM::parallel::printCICTimes(int world_rank){
    //MPI_Barrier(MPI_COMM_WORLD);
    CPUTimer_t total_min,total_max,total_mean,gpu_min,gpu_max,gpu_mean;
    HACCGPM::parallel::timing_stats(CIC_TIME,&total_min,&total_max,&total_mean);
    HACCGPM::parallel::timing_stats(CIC_KERNEL_TIME,&gpu_min,&gpu_max,&gpu_mean);
    //HACCGPM::parallel::timing_stats(TRANSFER_MPI_TIME,&mpi_min,&mpi_max,&mpi_mean);
    if (world_rank != 0)return;
    printf("   CIC                -> calls: %d\n",CIC_CALLS);
    printf("                               total: %10llu us mean | %10llu us max  | %10llu us min  |\n",total_mean,total_max,total_min);
    printf("                                 cpu: %10llu us mean | %10llu us max  | %10llu us min  |\n",(total_mean-gpu_mean),(total_max - gpu_max), (total_min - gpu_min));
    printf("                                 gpu: %10llu us mean | %10llu us max  | %10llu us min  |\n",gpu_mean,gpu_max,gpu_min);
    //printf("                                 mpi: %10llu us mean | %10llu us max  | %10llu us min  |\n",mpi_mean,mpi_max,mpi_min);
    printf("                                 avg: %10llu us mean | %10llu us max  | %10llu us min  |\n",total_mean / CIC_CALLS,total_max / CIC_CALLS,total_min / CIC_CALLS);
}

void HACCGPM::serial::CIC(deviceFFT_t* d_grid, float4* d_pos, int ng, int blockSize, int calls){
    CPUTimer_t start = CPUTimer();
    int numBlocks = (ng*ng*ng)/blockSize;
    getIndent(calls);
    #ifdef VerboseUpdate
    printf("%sCIC (complex) was called with\n%s   blockSize %d\n%s   numBlocks %d\n",indent,indent,blockSize,indent,numBlocks);
    #endif
    cudaCall(hipMemset,d_grid,0,sizeof(deviceFFT_t)*ng*ng*ng);
    CIC_KERNEL_TIME += InvokeGPUKernel(CICKernel,numBlocks,blockSize,d_grid,d_pos,ng,1.0f);
    CPUTimer_t end = CPUTimer();
    CPUTimer_t t = end-start;
    printf("%s   CIC (complex) took %llu us\n",indent,t);
    CIC_TIME += t;
    CIC_CALLS += 1;
}

void HACCGPM::serial::CIC(float* d_grid, float4* d_pos, int ng, int blockSize, int calls){
    CPUTimer_t start = CPUTimer();
    int numBlocks = (ng*ng*ng)/blockSize;
    getIndent(calls);
    #ifdef VerboseUpdate
    printf("%sCIC (float) was called with\n%s   blockSize %d\n%s   numBlocks %d\n",indent,indent,blockSize,indent,numBlocks);
    #endif
    cudaCall(hipMemset,d_grid,0,sizeof(float)*ng*ng*ng);
    CIC_KERNEL_TIME += InvokeGPUKernel(CICKernel,numBlocks,blockSize,d_grid,d_pos,ng,1.0f);
    CPUTimer_t end = CPUTimer();
    CPUTimer_t t = end-start;
    printf("%s   CIC (float) took %llu us\n",indent,t);
    CIC_TIME += t;
    CIC_CALLS += 1;
}

void HACCGPM::serial::CIC(deviceFFT_t* d_grid, float* d_temp, float4* d_pos, int ng, int blockSize, int calls){
    CPUTimer_t start = CPUTimer();
    int numBlocks = (ng*ng*ng)/blockSize;
    getIndent(calls);
    #ifdef VerboseUpdate
    printf("%sCIC (complex,float) was called with\n%s   blockSize %d\n%s   numBlocks %d\n",indent,indent,blockSize,indent,numBlocks);
    #endif
    cudaCall(hipMemset,d_temp,0,sizeof(float)*ng*ng*ng);
    CIC_KERNEL_TIME += InvokeGPUKernel(CICKernel,numBlocks,blockSize,d_temp,d_pos,ng,1.0f);
    CIC_KERNEL_TIME += InvokeGPUKernel(float2complex,numBlocks,blockSize,d_grid,d_temp,ng*ng*ng);
    CPUTimer_t end = CPUTimer();
    CPUTimer_t t = end-start;
    printf("%s   CIC (complex,float) took %llu us\n",indent,t);
    CIC_TIME += t;
    CIC_CALLS += 1;
}

void HACCGPM::serial::UpdateVelocities(float4* d_vel, float4* d_grad, float4* d_pos, HACCGPM::Timestepper ts, int ng, int blockSize, int calls){
    CPUTimer_t start = CPUTimer();
    int numBlocks = (ng*ng*ng)/blockSize;
    getIndent(calls);
    #ifdef VerboseUpdate
    printf("%sUpdate Velocities was called with\n%s   blockSize %d\n%s   numBlocks %d\n",indent,indent,blockSize,indent,numBlocks);
    #endif
    UPDATE_VEL_KERNEL_TIME += InvokeGPUKernel(ICICKernel,numBlocks,blockSize,d_vel,d_grad,d_pos,ts.deltaT,ts.fscal,ng);
    CPUTimer_t end = CPUTimer();
    CPUTimer_t t = end-start;
    printf("%s   UpdateVelocities took %llu us\n",indent,t);
    UPDATE_VEL_TIME += t;
    UPDATE_VEL_CALLS += 1;
}

void HACCGPM::serial::UpdatePositions(float4* d_pos, float4* d_vel, HACCGPM::Timestepper ts, float frac, int ng, int blockSize, int calls){
    CPUTimer_t start = CPUTimer();
    int numBlocks = (ng*ng*ng)/blockSize;
    float prefactor = ((ts.deltaT)/(ts.aa * ts.aa * ts.adot)) * frac;
    getIndent(calls);
    #ifdef VerboseUpdate
    printf("%sUpdate Positions was called with\n%s   blockSize %d\n%s   numBlocks %d\n%s   frac %g\n%s   prefactor %g\n",indent,indent,blockSize,indent,numBlocks,indent,frac,indent,prefactor);
    #endif
    UPDATE_POS_KERNEL_TIME += InvokeGPUKernel(UpdatePosKernel,numBlocks,blockSize,d_pos,d_vel,prefactor,(float)ng);
    CPUTimer_t end = CPUTimer();
    CPUTimer_t t = end-start;
    printf("%s   UpdatePositions took %llu us\n",indent,t);
    UPDATE_POS_TIME += t;
    UPDATE_POS_CALLS += 1;
}

void HACCGPM::serial::printCICTimes(){
    printf("   CIC               -> calls: %10d | total: %10llu us | cpu: %10llu us | gpu: %10llu us | mean: %10.2f us\n",CIC_CALLS,CIC_TIME,CIC_TIME - CIC_KERNEL_TIME,CIC_KERNEL_TIME,((float)CIC_TIME)/((float)(CIC_CALLS)));
    printf("   UpdatePositions   -> calls: %10d | total: %10llu us | cpu: %10llu us | gpu: %10llu us | mean: %10.2f us\n",UPDATE_POS_CALLS,UPDATE_POS_TIME,UPDATE_POS_TIME - UPDATE_POS_KERNEL_TIME,UPDATE_POS_KERNEL_TIME,((float)UPDATE_POS_TIME)/((float)(UPDATE_POS_CALLS)));
    printf("   UpdateVelocities  -> calls: %10d | total: %10llu us | cpu: %10llu us | gpu: %10llu us | mean: %10.2f us\n",UPDATE_VEL_CALLS,UPDATE_VEL_TIME,UPDATE_VEL_TIME - UPDATE_VEL_KERNEL_TIME,UPDATE_VEL_KERNEL_TIME,((float)UPDATE_VEL_TIME)/((float)(UPDATE_VEL_CALLS)));
}