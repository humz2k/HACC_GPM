#include "hip/hip_runtime.h"
#include "../ic_kernels.hpp"

template<class T>
__global__ void transformDensityField(const T* __restrict oldGrid, T* __restrict outSx, T* __restrict outSy, T* __restrict outSz, double delta, double rl, double a, int ng){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    double d = (2*M_PI)/rl;

    int3 idx3d = HACCGPM::serial::get_index(idx,ng);
    float3 kmodes = HACCGPM::get_kmodes(idx3d,ng,d);

    double k2 = kmodes.x * kmodes.x + kmodes.y * kmodes.y + kmodes.z * kmodes.z;

    double k2mul = (1/k2);
    if (k2 == 0){
        k2mul = 0;
    }

    double mul = (1/delta) * k2mul;

    T current = __ldg(&oldGrid[idx]);
    current.x *= mul;
    current.y *= mul;

    T sx,sy,sz;

    sx.x = current.y * kmodes.x;
    sx.y = -current.x * kmodes.x;

    sy.x = current.y * kmodes.y;
    sy.y = -current.x * kmodes.y;

    sz.x = current.y * kmodes.z;
    sz.y = -current.x * kmodes.z;

    outSx[idx] = sx;
    outSy[idx] = sy;
    outSz[idx] = sz;

}

__global__ void transformDensityField(const deviceFFT_t* __restrict oldGrid, deviceFFT_t* __restrict outSx, deviceFFT_t* __restrict outSy, deviceFFT_t* __restrict outSz, double delta, double rl, double a, int ng, int nlocal, int world_rank, int3 local_grid_size, int3 local_coords, int3 dims){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx >= nlocal)return;

    double d = (2*M_PI)/rl;

    int3 idx3d = HACCGPM::parallel::get_global_index(idx,ng,local_grid_size,local_coords);
    float3 kmodes = HACCGPM::get_kmodes(idx3d,ng,d);

    double k2 = kmodes.x * kmodes.x + kmodes.y * kmodes.y + kmodes.z * kmodes.z;

    double k2mul = (1/k2);
    if (k2 == 0){
        k2mul = 0;
    }

    double mul = (1/delta) * k2mul;

    deviceFFT_t current = __ldg(&oldGrid[idx]);
    current.x *= mul;
    current.y *= mul;

    deviceFFT_t sx,sy,sz;

    sx.x = current.y * kmodes.x;
    sx.y = -current.x * kmodes.x;

    sy.x = current.y * kmodes.y;
    sy.y = -current.x * kmodes.y;

    sz.x = current.y * kmodes.z;
    sz.y = -current.x * kmodes.z;

    outSx[idx] = sx;
    outSy[idx] = sy;
    outSz[idx] = sz;

}

void launch_transform_density_field(deviceFFT_t* d_grid, deviceFFT_t* d_x, deviceFFT_t* d_y, deviceFFT_t* d_z, double delta, double rl, double z_ini, int ng, int numBlocks, int blockSize, int calls){
    getIndent(calls);
    InvokeGPUKernel(transformDensityField,numBlocks,blockSize,d_grid,d_x,d_y,d_z,delta,rl,1/(1+z_ini),ng);
}

void launch_transform_density_field(floatFFT_t* d_grid, floatFFT_t* d_x, floatFFT_t* d_y, floatFFT_t* d_z, double delta, double rl, double z_ini, int ng, int numBlocks, int blockSize, int calls){
    getIndent(calls);
    InvokeGPUKernel(transformDensityField,numBlocks,blockSize,d_grid,d_x,d_y,d_z,delta,rl,1/(1+z_ini),ng);
}

void launch_transform_density_field(deviceFFT_t* d_grid, deviceFFT_t* d_x, deviceFFT_t* d_y, deviceFFT_t* d_z, double delta, double rl, double z_ini, int ng, int nlocal, int3 local_grid_size, int3 local_coords, int3 dims, int world_rank, int numBlocks, int blockSize, int calls){
    getIndent(calls);
    InvokeGPUKernelParallel(transformDensityField,numBlocks,blockSize,d_grid,d_x,d_y,d_z,delta,rl,1/(1+z_ini),ng,nlocal,world_rank,local_grid_size,local_coords,dims);
}