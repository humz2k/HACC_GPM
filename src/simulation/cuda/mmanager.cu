#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "haccgpm.hpp"

__global__ void set_invalid(float4* __restrict d_pos, int mem_frac){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    if (idx >= mem_frac) return;
    d_pos[idx] = make_float4(0,0,0,-10);
}

HACCGPM::parallel::MemoryManager::MemoryManager(HACCGPM::Params params){
    world_rank = params.world_rank;
    if (params.world_rank == 0)printf("MemoryManager:\n   Allocating d_vel,d_pos,d_greens,d_grid,d_x,d_y,d_z,d_grad,d_tempgrid...\n");
    int mem_frac = params.nlocal * params.frac;
    if (params.world_rank == 0)printf("   Mem frac: %d / %d\n",mem_frac,params.nlocal);
    cudaCall(hipMalloc,&d_vel,sizeof(float4)*mem_frac);
    if (params.world_rank == 0)printf("   Allocated d_vel: %lu bytes.\n",sizeof(float4)*mem_frac);

    cudaCall(hipMalloc,&d_pos,sizeof(float4)*mem_frac);
    if (params.world_rank == 0)printf("   Allocated d_pos: %lu bytes.\n",sizeof(float4)*mem_frac);

    int blockSize = params.blockSize;
    int numBlocks = (mem_frac + (blockSize - 1)) / blockSize;
    getIndent(0);
    InvokeGPUKernelParallel(set_invalid,numBlocks,blockSize,d_pos,mem_frac);
    if (params.world_rank == 0)printf("   set_invalid d_pos\n");

    cudaCall(hipMalloc,&d_greens,sizeof(hostFFT_t)*params.nlocal);
    if (params.world_rank == 0)printf("   Allocated d_greens: %lu bytes.\n",sizeof(hostFFT_t)*params.nlocal);

    cudaCall(hipMalloc,&d_grid,sizeof(deviceFFT_t)*params.nlocal);
    if (params.world_rank == 0)printf("   Allocated d_grid: %lu bytes.\n",sizeof(deviceFFT_t)*params.nlocal);

    cudaCall(hipMalloc,&d_x,sizeof(deviceFFT_t)*params.nlocal);
    if (params.world_rank == 0)printf("   Allocated d_x: %lu bytes.\n",sizeof(deviceFFT_t)*params.nlocal);

    cudaCall(hipMalloc,&d_y,sizeof(deviceFFT_t)*params.nlocal);
    if (params.world_rank == 0)printf("   Allocated d_y: %lu bytes.\n",sizeof(deviceFFT_t)*params.nlocal);

    cudaCall(hipMalloc,&d_z,sizeof(deviceFFT_t)*params.nlocal);
    if (params.world_rank == 0)printf("   Allocated d_z: %lu bytes.\n",sizeof(deviceFFT_t)*params.nlocal);

    double ol = params.ol;
    if (params.world_rank == 0)printf("   OL (mpc) = %g\n",ol);
    int overload = params.overload;
    if (params.world_rank == 0)printf("   overload (grid) = %d\n",overload);

    int x = params.local_grid_size[0] + overload*2;
    int y = params.local_grid_size[1] + overload*2;
    int z = params.local_grid_size[2] + overload*2;

    if (params.world_rank == 0)printf("   overload volume = (%d %d %d) + 2*(%d %d %d) = (%d %d %d)\n",params.local_grid_size[0],
                                                                                    params.local_grid_size[1],
                                                                                    params.local_grid_size[2],
                                                                                    overload,
                                                                                    overload,
                                                                                    overload,
                                                                                    x,y,z);

    int extra_grid_size = (x)*(y)*(z);
    cudaCall(hipMalloc,&d_tempgrid,sizeof(float)*extra_grid_size);
    if (params.world_rank == 0)printf("   Allocated d_tempgrid: %lu bytes.\n",sizeof(float)*extra_grid_size);

    //cudaCall(hipMalloc,&d_extragrid,sizeof(deviceFFT_t)*(params.ng));

    //cudaCall(hipMalloc,&d_grid2,sizeof(deviceFFT_t)*params.nlocal);
    //if (params.world_rank == 0)printf("   Allocated d_grid: %lu bytes.\n",sizeof(deviceFFT_t)*params.nlocal);

    cudaCall(hipMalloc,&d_grad,sizeof(float4)*extra_grid_size);
    if (params.world_rank == 0)printf("   Allocated d_grad: %lu bytes.\n",sizeof(float4)*extra_grid_size);
}

HACCGPM::parallel::MemoryManager::~MemoryManager(){
    if (world_rank == 0)printf("MemoryManager:\n   Freeing d_vel,d_pos,d_greens,d_grid,d_x,d_y,d_z,d_grad,d_tempgrid...\n");
    
    cudaCall(hipFree,d_pos);
    cudaCall(hipFree,d_vel);
    cudaCall(hipFree,d_greens);
    cudaCall(hipFree,d_grid);
    cudaCall(hipFree,d_tempgrid);
    cudaCall(hipFree,d_grad);
    cudaCall(hipFree,d_x);
    cudaCall(hipFree,d_y);
    cudaCall(hipFree,d_z);

    if (world_rank == 0)printf("      Freed d_vel,d_pos,d_greens,d_grid,d_x,d_y,d_z,d_grad,d_tempgrid.\n");
}

HACCGPM::serial::MemoryManager::MemoryManager(HACCGPM::Params params){
    #ifdef USE_TEMP_GRID
    #ifdef USE_ONE_GRID
    printf("MemoryManager:\n   Allocating d_vel,d_pos,d_greens,d_grid,d_tempgrid,d_grad...\n");
    #else
    printf("MemoryManager:\n   Allocating d_vel,d_pos,d_greens,d_grid,d_x,d_y,d_z,d_tempgrid,d_grad...\n");
    #endif
    #else
    #ifdef USE_ONE_GRID
    printf("MemoryManager:\n   Allocating d_vel,d_pos,d_greens,d_grid,d_grad...\n");
    #else
    printf("MemoryManager:\n   Allocating d_vel,d_pos,d_greens,d_grid,d_x,d_y,d_z,d_grad...\n");
    #endif
    #endif
    
    size_t total_memory = 0;

    cudaCall(hipMalloc,&d_vel,sizeof(particle_t)*params.ng*params.ng*params.ng);
    printf("   Allocated d_vel: %lu bytes.\n",sizeof(particle_t)*params.ng*params.ng*params.ng);

    total_memory += sizeof(particle_t)*params.ng*params.ng*params.ng;

    cudaCall(hipMalloc,&d_pos,sizeof(particle_t)*params.ng*params.ng*params.ng);
    printf("   Allocated d_pos: %lu bytes.\n",sizeof(particle_t)*params.ng*params.ng*params.ng);

    total_memory += sizeof(particle_t)*params.ng*params.ng*params.ng;

    cudaCall(hipMalloc,&d_greens,sizeof(greens_t)*params.ng*params.ng*params.ng);
    printf("   Allocated d_greens: %lu bytes.\n",sizeof(greens_t)*params.ng*params.ng*params.ng);

    total_memory += sizeof(greens_t)*params.ng*params.ng*params.ng;

    cudaCall(hipMalloc,&d_grid,sizeof(grid_t)*params.ng*params.ng*params.ng);
    printf("   Allocated d_grid: %lu bytes.\n",sizeof(grid_t)*params.ng*params.ng*params.ng);

    total_memory += sizeof(grid_t)*params.ng*params.ng*params.ng;

    #ifndef USE_ONE_GRID
    cudaCall(hipMalloc,&d_x,sizeof(grid_t)*params.ng*params.ng*params.ng);
    printf("   Allocated d_x: %lu bytes.\n",sizeof(grid_t)*params.ng*params.ng*params.ng);

    total_memory += sizeof(grid_t)*params.ng*params.ng*params.ng;

    cudaCall(hipMalloc,&d_y,sizeof(grid_t)*params.ng*params.ng*params.ng);
    printf("   Allocated d_y: %lu bytes.\n",sizeof(grid_t)*params.ng*params.ng*params.ng);

    total_memory += sizeof(grid_t)*params.ng*params.ng*params.ng;

    cudaCall(hipMalloc,&d_z,sizeof(grid_t)*params.ng*params.ng*params.ng);
    printf("   Allocated d_z: %lu bytes.\n",sizeof(grid_t)*params.ng*params.ng*params.ng);

    total_memory += sizeof(grid_t)*params.ng*params.ng*params.ng;
    #endif

    #ifdef USE_TEMP_GRID
    cudaCall(hipMalloc,&d_tempgrid,sizeof(float)*params.ng*params.ng*params.ng);
    printf("   Allocated d_tempgrid: %lu bytes.\n",sizeof(float)*params.ng*params.ng*params.ng);

    total_memory += sizeof(float)*params.ng*params.ng*params.ng;
    #endif

    cudaCall(hipMalloc,&d_grad,sizeof(float4)*params.ng*params.ng*params.ng);
    printf("   Allocated d_grad: %lu bytes.\n",sizeof(float4)*params.ng*params.ng*params.ng);

    total_memory += sizeof(float4)*params.ng*params.ng*params.ng;

    printf("Total: %g GB\n",((double)total_memory) * 1e-9);
}

HACCGPM::serial::MemoryManager::~MemoryManager(){
    #ifdef USE_TEMP_GRID
    #ifdef USE_ONE_GRID
    printf("MemoryManager:\n   Freeing d_vel,d_pos,d_greens,d_grid,d_tempgrid,d_grad...\n");
    #else
    printf("MemoryManager:\n   Freeing d_vel,d_pos,d_greens,d_grid,d_x,d_y,d_z,d_tempgrid,d_grad...\n");
    #endif
    #else
    #ifdef USE_ONE_GRID
    printf("MemoryManager:\n   Freeing d_vel,d_pos,d_greens,d_grid,d_grad...\n");
    #else
    printf("MemoryManager:\n   Freeing d_vel,d_pos,d_greens,d_grid,d_x,d_y,d_z,d_grad...\n");
    #endif
    #endif
    
    cudaCall(hipFree,d_pos);
    cudaCall(hipFree,d_vel);
    cudaCall(hipFree,d_greens);
    cudaCall(hipFree,d_grid);
    #ifdef USE_TEMP_GRID
    cudaCall(hipFree,d_tempgrid);
    #endif
    cudaCall(hipFree,d_grad);
    #ifndef USE_ONE_GRID
    cudaCall(hipFree,d_x);
    cudaCall(hipFree,d_y);
    cudaCall(hipFree,d_z);
    #endif

    #ifdef USE_TEMP_GRID
    #ifdef USE_ONE_GRID
    printf("      Freed d_vel,d_pos,d_greens,d_grid,d_temp_grid,d_grad.\n");
    #else
    printf("      Freed d_vel,d_pos,d_greens,d_grid,d_x,d_y,d_z,d_temp_grid,d_grad.\n");
    #endif
    #else
    #ifdef USE_ONE_GRID
    printf("      Freed d_vel,d_pos,d_greens,d_grid,d_grad.\n");
    #else
    printf("      Freed d_vel,d_pos,d_greens,d_grid,d_x,d_y,d_z,d_grad.\n");
    #endif
    #endif
}