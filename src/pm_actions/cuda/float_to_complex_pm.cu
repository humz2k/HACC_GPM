#include "hip/hip_runtime.h"
#include "../pm_kernels.hpp"

__global__ void float2complex(deviceFFT_t* __restrict d_out, const float* __restrict d_in, int n){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    if(idx >= n)return;
    float my_grid = __ldg(&d_in[idx]);
    deviceFFT_t out;
    out.x = my_grid;
    out.y = 0;
    d_out[idx] = out;
}

__global__ void float2complex(deviceFFT_t* __restrict d_out, const float* __restrict d_in, int3 local_grid_size, int overload){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    int n = (local_grid_size.x + 2*overload)*(local_grid_size.y + 2*overload)*(local_grid_size.z + 2*overload);
    if(idx >= n)return;
    int3 ol_grid_size = make_int3(local_grid_size.x + 2*overload,local_grid_size.y + 2*overload,local_grid_size.z + 2*overload);
    int3 idx3d = HACCGPM::parallel::get_local_index(idx,ol_grid_size.x,ol_grid_size.y,ol_grid_size.z);

    idx3d.x -= overload;
    idx3d.y -= overload;
    idx3d.z -= overload;

    if (idx3d.x < 0)return;
    if (idx3d.y < 0)return;
    if (idx3d.z < 0)return;
    if (idx3d.x >= local_grid_size.x)return;
    if (idx3d.y >= local_grid_size.y)return;
    if (idx3d.z >= local_grid_size.z)return;

    float my_grid = __ldg(&d_in[idx]);
    deviceFFT_t out;
    out.x = my_grid;
    out.y = 0;
    int outidx = idx3d.x * local_grid_size.y * local_grid_size.z + idx3d.y * local_grid_size.z + idx3d.z;
    d_out[outidx] = out;
}

CPUTimer_t launch_f2c(deviceFFT_t* d_out, float* d_in, int ng, int numBlocks, int blockSize, int calls){
    getIndent(calls);
    return InvokeGPUKernel(float2complex,numBlocks,blockSize,d_out,d_in,ng*ng*ng);
}

CPUTimer_t launch_f2c(deviceFFT_t* d_out, float* d_in, int3 local_grid_size, int overload, int world_rank, int numBlocks, int blockSize, int calls){
    getIndent(calls);
    return InvokeGPUKernelParallel(float2complex,numBlocks,blockSize,d_out,d_in,local_grid_size,overload);
}