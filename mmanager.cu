#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "stdio.h"
#include "haccgpm.hpp"

HACCGPM::serial::MemoryManager::MemoryManager(HACCGPM::Params params){
    printf("MemoryManager:\n   Allocating d_vel,d_pos,d_greens,d_grid,d_grad...\n");
    
    cudaCall(hipMalloc,&d_vel,sizeof(float4)*params.ng*params.ng*params.ng);
    printf("   Allocated d_vel: %lu bytes.\n",sizeof(float4)*params.ng*params.ng*params.ng);

    cudaCall(hipMalloc,&d_pos,sizeof(float4)*params.ng*params.ng*params.ng);
    printf("   Allocated d_pos: %lu bytes.\n",sizeof(float4)*params.ng*params.ng*params.ng);

    cudaCall(hipMalloc,&d_greens,sizeof(hostFFT_t)*params.ng*params.ng*params.ng);
    printf("   Allocated d_greens: %lu bytes.\n",sizeof(hostFFT_t)*params.ng*params.ng*params.ng);

    cudaCall(hipMalloc,&d_grid,sizeof(deviceFFT_t)*params.ng*params.ng*params.ng);
    printf("   Allocated d_grid: %lu bytes.\n",sizeof(deviceFFT_t)*params.ng*params.ng*params.ng);

    cudaCall(hipMalloc,&d_grad,sizeof(float4)*params.ng*params.ng*params.ng);
    printf("   Allocated d_grad: %lu bytes.\n",sizeof(float4)*params.ng*params.ng*params.ng);
}

HACCGPM::serial::MemoryManager::~MemoryManager(){
    printf("MemoryManager:\n   Freeing d_vel,d_pos,d_greens,d_grid,d_grad...\n");
    
    cudaCall(hipFree,d_pos);
    cudaCall(hipFree,d_vel);
    cudaCall(hipFree,d_greens);
    cudaCall(hipFree,d_grid);
    cudaCall(hipFree,d_grad);

    printf("      Freed d_vel,d_pos,d_greens,d_grid,d_grad.\n");
}