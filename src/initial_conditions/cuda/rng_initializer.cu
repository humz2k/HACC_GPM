#include "hip/hip_runtime.h"
#include "../ic_kernels.hpp"

__global__ void initRNG(hiprandState *state, int seed){

  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(seed, idx, 0, &state[idx]);

}

__global__ void initRNG(hiprandState *state, int seed, int nlocal, int ng, int3 local_grid_size, int3 local_coords){

  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  if (idx >= nlocal)return;
  int3 idx3d = HACCGPM::parallel::get_global_index(idx,ng,local_grid_size,local_coords);
  int global_idx = idx3d.x * ng * ng + idx3d.y * ng + idx3d.z;
  hiprand_init(seed, global_idx, 0, &state[idx]);

}

__global__ void GenerateRealRandom(hiprandState* state, deviceFFT_t* __restrict grid, int nlocal){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    if (idx >= nlocal)return;
    hostFFT_t amp = hiprand_normal_double(state + idx);
    deviceFFT_t out;
    out.x = amp;
    out.y = 0;
    grid[idx] = out;
}

void launch_generate_rng(deviceFFT_t* d_grid1, int ng, int seed, int numBlocks, int blockSize, int calls){
    getIndent(calls);
    hiprandState* rngState; cudaCall(hipMalloc,&rngState,sizeof(hiprandState)*ng*ng*ng);
    InvokeGPUKernel(initRNG,numBlocks,blockSize,rngState,seed);
    InvokeGPUKernel(GenerateRealRandom,numBlocks,blockSize,rngState,d_grid1,ng*ng*ng);
    cudaCall(hipFree,rngState);
}